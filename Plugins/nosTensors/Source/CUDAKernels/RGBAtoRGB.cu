#include "hip/hip_runtime.h"

extern "C" {
	__global__ void RGBAtoRGB(void* InData, int TotalSizeInBytes, int BytesPerElement, int Width, int Height, void* OutData) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		char* inData = reinterpret_cast<char*>(InData);
		char* outData = reinterpret_cast<char*>(OutData);
		idx = idx * BytesPerElement;
		if ((idx + Width*Height*2*BytesPerElement + BytesPerElement) < (TotalSizeInBytes) && (idx*4 + 3*BytesPerElement) < TotalSizeInBytes) {
			for (int i = 0; i < BytesPerElement; i++) { //Copy all bytes!
								   				   *(outData + idx + i)	= *(inData + i + idx*4);				 	 //R
				   *(outData + (idx + Width*Height*BytesPerElement)+ i) = *(inData + i + idx*4 + 1*BytesPerElement); //G
				 *(outData + (idx + Width*Height*2*BytesPerElement)+ i)	= *(inData + i + idx*4 + 2*BytesPerElement); //B
			}
		}
	}
}