#include "hip/hip_runtime.h"
#include "CommonCUDAKernels.h"

__global__ void NormalizeKernel(float* data, float max, int dataSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize) {
        data[idx] = (data[idx])/max;
    }
}
    void NormalizeKernelWrapper(dim3 blocks, dim3 thread, float* data, float max, int dataSize)
    {
        NormalizeKernel<<<blocks, thread >>> (data, max, dataSize);
    }
