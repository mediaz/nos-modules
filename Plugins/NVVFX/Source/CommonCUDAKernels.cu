#include "hip/hip_runtime.h"



__global__ void NormalizeKernel(float* data, float max, int dataSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize) {
        data[idx] = (data[idx])/max;
    }
}